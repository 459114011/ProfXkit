#include "hip/hip_runtime.h"
// Please choose a data type to compile
#define DATATYPE 0

#if DATATYPE==0
    #pragma message "Compiling using StorageT=half ComputeT=float"
    #define StorageT half
    #define ComputeT float
    #define sizeofStorageT 2
    #define sizeofComputeT 4
    #define CUDNNStorageT HIPDNN_DATA_HALF
    #define CUDNNConvComputeT HIPDNN_DATA_FLOAT
    #define CPUStorage2ComputeT(x) (cpu_half2float(x))
    #define CPUCompute2StorageT(x) (cpu_float2half(x))
    #define GPUStorage2ComputeT(x) (__half2float(x))
    #define GPUCompute2StorageT(x) (__float2half(x))
    #define GPUgemm Hgemm
    #define GPUasum Hasum
    #define ISNAN(x) (ishnan(x))
    #define ComputeT_MIN FLT_MIN
#elif DATATYPE==1
    #pragma message "Compiling using StorageT=float ComputeT=float"
    #define StorageT float
    #define ComputeT float
    #define sizeofStorageT 4
    #define sizeofComputeT 4
    #define CUDNNStorageT HIPDNN_DATA_FLOAT
    #define CUDNNConvComputeT HIPDNN_DATA_FLOAT
    #define CPUStorage2ComputeT(x) (x)
    #define CPUCompute2StorageT(x) (x)
    #define GPUStorage2ComputeT(x) (x)
    #define GPUCompute2StorageT(x) (x)
    #define GPUgemm hipblasSgemm
    #define GPUasum hipblasSasum
    #define ISNAN(x) (std::isnan(x))
    #define ComputeT_MIN FLT_MIN
#elif DATATYPE==2
    #pragma message "Compiling using StorageT=double ComputeT=double"
    #define StorageT double
    #define ComputeT double
    #define sizeofStorageT 8
    #define sizeofComputeT 8
    #define CUDNNStorageT HIPDNN_DATA_DOUBLE
    #define CUDNNConvComputeT HIPDNN_DATA_DOUBLE
    #define CPUStorage2ComputeT(x) (x)
    #define CPUCompute2StorageT(x) (x)
    #define GPUStorage2ComputeT(x) (x)
    #define GPUCompute2StorageT(x) (x)
    #define GPUgemm hipblasDgemm
    #define GPUasum hipblasDasum
    #define ISNAN(x) (std::isnan(x))
    #define ComputeT_MIN DBL_MIN
#endif

//////////////////////////////////////////////////////////////////////////////////////////////////
// Includes
//////////////////////////////////////////////////////////////////////////////////////////////////
#include <hip/hip_fp16.h>
#include <cstdlib>
#include <cstdio>
#include <cstdarg>
#include <cmath>
#include <cfloat>
#include <iostream>
#include <fstream>
#include <sstream>
#include <random>
#include <algorithm>
#include <map>
#include <vector>
#include <string>
#include <typeinfo>
#include <typeindex>
#include <thread>
#include <chrono>
#include <future>
#include <sys/time.h>



//////////////////////////////////////////////////////////////////////////////////////////////////
// Debugging utility
//////////////////////////////////////////////////////////////////////////////////////////////////

void FatalError(const int lineNumber=0) {
    std::cerr << "FatalError";
    if (lineNumber!=0) std::cerr<<" at LINE "<<lineNumber;
    std::cerr << ". Program Terminated." << std::endl;
    hipDeviceReset();
    exit(EXIT_FAILURE);
}

void checkCUDA(const int lineNumber, hipError_t status) {
    if (status != hipSuccess) {
        std::cerr << "CUDA failure at LINE " << lineNumber << ": " << status << std::endl;
        FatalError();
    }
}

//////////////////////////////////////////////////////////////////////////////////////////////////
// HALF computation ultility
//////////////////////////////////////////////////////////////////////////////////////////////////

static __inline__ __device__ __host__ int ishnan(half h) {
    // When input is NaN, exponent is all ones and mantissa is non-zero.
    return (h.x & 0x7c00U) == 0x7c00U && (h.x & 0x03ffU) != 0;
}

half cpu_float2half(float f) {
    half ret;

    unsigned x = *((int*)(void*)(&f));
    unsigned u = (x & 0x7fffffff), remainder, shift, lsb, lsb_s1, lsb_m1;
    unsigned sign, exponent, mantissa;

    // Get rid of +NaN/-NaN case first.
    if (u > 0x7f800000) {
        ret.x = 0x7fffU;
        return ret;
    }

    sign = ((x >> 16) & 0x8000);

    // Get rid of +Inf/-Inf, +0/-0.
    if (u > 0x477fefff) {
        ret.x = sign | 0x7c00U;
        return ret;
    }
    if (u < 0x33000001) {
        ret.x = (sign | 0x0000);
        return ret;
    }

    exponent = ((u >> 23) & 0xff);
    mantissa = (u & 0x7fffff);

    if (exponent > 0x70) {
        shift = 13;
        exponent -= 0x70;
    } else {
        shift = 0x7e - exponent;
        exponent = 0;
        mantissa |= 0x800000;
    }
    lsb = (1 << shift);
    lsb_s1 = (lsb >> 1);
    lsb_m1 = (lsb - 1);

    // Round to nearest even.
    remainder = (mantissa & lsb_m1);
    mantissa >>= shift;
    if (remainder > lsb_s1 || (remainder == lsb_s1 && (mantissa & 0x1))) {
        ++mantissa;
        if (!(mantissa & 0x3ff)) {
            ++exponent;
            mantissa = 0;
        }
    }

    ret.x = (sign | (exponent << 10) | mantissa);

    return ret;
}


float cpu_half2float(half h) {
    unsigned sign = ((h.x >> 15) & 1);
    unsigned exponent = ((h.x >> 10) & 0x1f);
    unsigned mantissa = ((h.x & 0x3ff) << 13);

    if (exponent == 0x1f) {  /* NaN or Inf */
        mantissa = (mantissa ? (sign = 0, 0x7fffff) : 0);
        exponent = 0xff;
    } else if (!exponent) {  /* Denorm or Zero */
        if (mantissa) {
            unsigned int msb;
            exponent = 0x71;
            do {
                msb = (mantissa & 0x400000);
                mantissa <<= 1;  /* normalize */
                --exponent;
            } while (!msb);
            mantissa &= 0x7fffff;  /* 1.mantissa is implicit */
        }
    } else {
        exponent += 0x70;
    }

    int temp = ((sign << 31) | (exponent << 23) | mantissa);

    return *((float*)((void*)&temp));
}


bool operator <(const half& x, const half& y) {
    return cpu_half2float(x) < cpu_half2float(y);
}

std::ostream& operator<< (std::ostream& stream, const half& x) {
    stream << cpu_half2float(x);
    return stream;
}


//////////////////////////////////////////////////////////////////////////////////////////////////
// File format
//////////////////////////////////////////////////////////////////////////////////////////////////

void memorySizePrint(size_t bytes){
    if (bytes<512){
        std::cout<<bytes<<" Bytes";
    }else if (bytes<512.0*1024){
        std::cout<<(bytes/1024.0)<<" KB";
    }else if (bytes<512.0*1024*1024){
        std::cout<<(bytes/(1024.0*1024.0))<<" MB";
    }else if (bytes<512.0*1024*1024*1024){
        std::cout<<(bytes/(1024.0*1024.0*1024.0))<<" GB";
    }else if (bytes<512.0*1024*1024*1024*1024){
        std::cout<<(bytes/(1024.0*1024.0*1024.0*1024.0))<<" TB";
    }else{
        std::cout<<(bytes/(1024.0*1024.0*1024.0*1024.0*1024.0))<<" PB";
    }
}

void veciPrint(const std::vector<int>& v){
    std::cout<<"["<<v.size()<<"]={";
    if (v.size()>0) std::cout<<v[0];
    if (v.size()>1){
        for (int i=1;i<v.size();++i){
            std::cout<<","<<v[i];
        }
    }
    std::cout<<"}";
}

size_t numel(const std::vector<int>& dim){
    size_t res = 1;
    for (int i=0;i<dim.size();++i) res *= (size_t)(dim[i]);
    return res;
}

size_t sizeofitem(const std::vector<int>& dim){
    size_t res = 1;
    for (int i=1;i<dim.size();++i) res *= (size_t)(dim[i]);
    return res;
}

size_t numspel(const std::vector<int>& dim){
    size_t res = 1;
    for (int i=2;i<dim.size();++i) res *= (size_t)(dim[i]);
    return res;
}

uint8_t typeID(std::type_index t){
    if (t==typeid(half))        return uint8_t(0);
    if (t==typeid(float))       return uint8_t(1);
    if (t==typeid(double))      return uint8_t(2);
    if (t==typeid(uint8_t))     return uint8_t(3);
    if (t==typeid(uint16_t))    return uint8_t(4);
    if (t==typeid(uint32_t))    return uint8_t(5);
    if (t==typeid(uint64_t))    return uint8_t(6);
    if (t==typeid(int8_t))      return uint8_t(7);
    if (t==typeid(int16_t))     return uint8_t(8);
    if (t==typeid(int32_t))     return uint8_t(9);
    if (t==typeid(int64_t))     return uint8_t(10);
    if (t==typeid(char))        return uint8_t(11);
    if (t==typeid(bool))        return uint8_t(12);
    FatalError(__LINE__);       return uint8_t(255);
}

uint8_t readTypeID(std::string filename){
    FILE* fp = fopen(filename.c_str(),"rb");
    while (fp==NULL) {
        std::cerr<<"readTypeID: fail to open file "<<filename<<". Please provide it first. Will retry after 5 seconds."<<std::endl;
        std::this_thread::sleep_for(std::chrono::seconds(5));
        fp = fopen(filename.c_str(),"rb");
    }
    size_t read_cnt;
    uint8_t fpTypeid; read_cnt = fread((void*)(&fpTypeid), sizeof(uint8_t), 1, fp);     if (read_cnt!=1) { std::cerr<<"Error at readTypeID: no data type. "<<std::endl; FatalError(__LINE__); }
    fclose(fp);
    return fpTypeid;
}

template <class T>
class Tensor{
public:
    std::vector<int> dim;
    T* CPUmem;
    std::string name;

    // compile will check if your time is not correct for writeGPU and readGPU
    void writeGPU(T* GPUmem){
        hipMemcpy(GPUmem, CPUmem, numel()*sizeof(T), hipMemcpyHostToDevice);
    };

    void readGPU(T* GPUmem){
        hipMemcpy(CPUmem, GPUmem, numel()*sizeof(T), hipMemcpyDeviceToHost);
    };

    Tensor(): CPUmem(NULL){};

    size_t numel(){ return ::numel(dim); };

    size_t numBytes(){ return sizeof(T)*numel(); };

    int numofitems(){ return dim[0]; };

    size_t sizeofitem(){ return ::sizeofitem(dim); };

    ~Tensor(){
        if (CPUmem!=NULL)   delete[] CPUmem;
    };

    void initialize(T val){
        for (size_t i=0;i<numel();++i){
            CPUmem[i]=val;
        }
    };

    size_t readHeader(FILE* fp){
        size_t read_cnt;
        uint8_t myTypeid = typeID(typeid(T));
        uint32_t myTypesizeof = uint32_t(sizeof(T));
        uint8_t fpTypeid;       read_cnt = fread((void*)(&fpTypeid), sizeof(uint8_t), 1, fp);       if (read_cnt!=1) { std::cerr<<"Error at Tensor::readHeader: no data type. "<<std::endl; FatalError(__LINE__); }
        uint32_t fpTypesizeof;  read_cnt = fread((void*)(&fpTypesizeof), sizeof(uint32_t), 1, fp);  if (read_cnt!=1) { std::cerr<<"Error at Tensor::readHeader: no data size. "<<std::endl; FatalError(__LINE__); }
        int lenName;
        read_cnt = fread((void*)(&lenName), sizeof(int), 1, fp);
        if (read_cnt!=1) { std::cerr<<"Error at Tensor::readHeader: wrong data type. "<<std::endl; FatalError(__LINE__); }
        name.resize(lenName);
        if (lenName>0){
            read_cnt = fread((void*)(name.data()), sizeof(char), lenName, fp);
            if (read_cnt!=lenName) { std::cerr<<"Error at Tensor::readHeader: wrong data type. "<<std::endl; FatalError(__LINE__); }
        }
        int nbDims;
        read_cnt = fread((void*)(&nbDims), sizeof(int), 1, fp);
        if (read_cnt!=1) { std::cerr<<"Error at Tensor::readHeader: wrong data type. "<<std::endl; FatalError(__LINE__); }
        dim.resize(nbDims);
        if (nbDims>0){
            read_cnt = fread((void*)(&dim[0]), sizeof(int), nbDims, fp);
            if (read_cnt!=nbDims) { std::cerr<<"Error at Tensor::readHeader: wrong data type. "<<std::endl; FatalError(__LINE__); }
        }

        size_t headerBytes = sizeof(uint8_t) + sizeof(uint32_t) + sizeof(int) + lenName*sizeof(char) + sizeof(int) + nbDims*sizeof(int);

        if (myTypeid!=fpTypeid || myTypesizeof!=fpTypesizeof){
            std::cerr<<"Error at Tensor::readHeader: wrong data type. "<<std::endl; FatalError(__LINE__);
        }

        return headerBytes;
    };

    //support continuous read across many NdTensors
    T* read(FILE* fp,int batch_size=1){
        if (CPUmem!=NULL){
            delete[] CPUmem;
            CPUmem = NULL;
        }

        size_t read_cnt;

        uint8_t myTypeid = typeID(typeid(T));
        uint32_t myTypesizeof = uint32_t(sizeof(T));

        uint8_t fpTypeid;       read_cnt = fread((void*)(&fpTypeid), sizeof(uint8_t), 1, fp);       if (read_cnt!=1) return NULL;
        uint32_t fpTypesizeof;  read_cnt = fread((void*)(&fpTypesizeof), sizeof(uint32_t), 1, fp);  if (read_cnt!=1) return NULL;

        if (myTypeid!=fpTypeid || myTypesizeof!=fpTypesizeof){

            if (myTypeid==fpTypeid && myTypesizeof!=fpTypesizeof){ std::cerr<<"Tensor read error: same type but different sizeof, maybe different computer architecture. "<<std::endl; FatalError(__LINE__);}

            //if (myTypeid!=fpTypeid){ std::cerr<<"Tensor read error: different types. "<<std::endl; FatalError(__LINE__); }

            if (myTypeid==typeID(typeid(half)) && fpTypeid==typeID(typeid(float))){
                //std::cout<<std::endl<<"converting from float to half"<<std::endl;
                fseek(fp, -(sizeof(uint8_t)+sizeof(uint32_t)), SEEK_CUR);
                Tensor<float>* floatTensor = new Tensor<float>(fp);
                this->dim  = floatTensor->dim ;
                this->name = floatTensor->name;
                Malloc(batch_size);
                for(size_t i=0; i<numel(); ++i){
                    half v = cpu_float2half(floatTensor->CPUmem[i]);
                    memcpy(((half*)(CPUmem))+i,&v,sizeof(half));
                }
                delete floatTensor;
            }else if (myTypeid==typeID(typeid(float)) && fpTypeid==typeID(typeid(half))){
                fseek(fp, -(sizeof(uint8_t)+sizeof(uint32_t)), SEEK_CUR);
                Tensor<half>* halfTensor = new Tensor<half>(fp);
                this->dim  = halfTensor->dim ;
                this->name = halfTensor->name;
                Malloc(batch_size);
                for(size_t i=0; i<numel(); ++i){
                    float v = cpu_half2float(halfTensor->CPUmem[i]);
                    memcpy(((float*)(CPUmem))+i,&v,sizeof(float));
                }
                delete halfTensor;
            }else if (myTypeid==typeID(typeid(double)) && fpTypeid==typeID(typeid(float))){
                fseek(fp, -(sizeof(uint8_t)+sizeof(uint32_t)), SEEK_CUR);
                Tensor<float>* floatTensor = new Tensor<float>(fp);
                this->dim  = floatTensor->dim ;
                this->name = floatTensor->name;
                Malloc(batch_size);
                for(size_t i=0; i<numel(); ++i){
                    double v = double(floatTensor->CPUmem[i]);
                    memcpy(((double*)(CPUmem))+i,&v,sizeof(double));
                }
                delete floatTensor;
            }else if (myTypeid==typeID(typeid(float)) && fpTypeid==typeID(typeid(double))){
                fseek(fp, -(sizeof(uint8_t)+sizeof(uint32_t)), SEEK_CUR);
                Tensor<double>* doubleTensor = new Tensor<double>(fp);
                this->dim  = doubleTensor->dim ;
                this->name = doubleTensor->name;
                Malloc(batch_size);
                for(size_t i=0; i<numel(); ++i){
                    float v = float(doubleTensor->CPUmem[i]);
                    memcpy(((float*)(CPUmem))+i,&v,sizeof(float));
                }
                delete doubleTensor;
            }else if (myTypeid==typeID(typeid(half)) && fpTypeid==typeID(typeid(double))){
                fseek(fp, -(sizeof(uint8_t)+sizeof(uint32_t)), SEEK_CUR);
                Tensor<double>* doubleTensor = new Tensor<double>(fp);
                this->dim  = doubleTensor->dim ;
                this->name = doubleTensor->name;
                Malloc(batch_size);
                for(size_t i=0; i<numel(); ++i){
                    half v = cpu_float2half(float(doubleTensor->CPUmem[i]));
                    memcpy(((half*)(CPUmem))+i,&v,sizeof(half));
                }
                delete doubleTensor;
            }else if (myTypeid==typeID(typeid(float)) && fpTypeid==typeID(typeid(half))){
                fseek(fp, -(sizeof(uint8_t)+sizeof(uint32_t)), SEEK_CUR);
                Tensor<half>* halfTensor = new Tensor<half>(fp);
                this->dim  = halfTensor->dim ;
                this->name = halfTensor->name;
                Malloc(batch_size);
                for(size_t i=0; i<numel(); ++i){
                    double v = double(cpu_half2float(halfTensor->CPUmem[i]));
                    memcpy(((double*)(CPUmem))+i,&v,sizeof(double));
                }
                delete halfTensor;
            }else{
                std::cerr<<"Tensor conversion is not supported: from Type "<<fpTypeid<<" to Type "<<myTypeid<<std::endl;
                FatalError(__LINE__);
            }

        }else{
            int lenName;
            read_cnt = fread((void*)(&lenName), sizeof(int), 1, fp);
            if (read_cnt!=1) return NULL;
            name.resize(lenName);
            if (lenName>0){
                read_cnt = fread((void*)(name.data()), sizeof(char), lenName, fp);
                if (read_cnt!=lenName) return NULL;
            }
            int nbDims;
            read_cnt = fread((void*)(&nbDims), sizeof(int), 1, fp);
            if (read_cnt!=1) return NULL;
            dim.resize(nbDims);
            if (nbDims>0){
                read_cnt = fread((void*)(&dim[0]), sizeof(int), nbDims, fp);
                if (read_cnt!=nbDims) return NULL;
            }

            size_t n = numel();
            Malloc(batch_size);
            read_cnt = fread((void*)(CPUmem), sizeof(T), n, fp);
            if (read_cnt!=n){
                delete [] CPUmem;
                CPUmem = NULL;
                return NULL;
            }
        }

        return CPUmem;
    };

    void Malloc(int batch_size){
        size_t n = numel();
        //std::cout<<"  ";        memorySizePrint(n*sizeof(T));   std::cout<<std::endl;

        if (batch_size==1 || dim[0]%batch_size ==0){
            CPUmem = new T [n];
        }else{
            int dim0 =  (dim[0]/batch_size + 1) * batch_size;
            size_t oversize = n/dim[0] * dim0;
            CPUmem = new T [oversize];
            memset((void*)(CPUmem+n),0, (oversize-n)*sizeof(T));
        }
    };

    T* read(std::string filename,int batch_size=1){
        FILE* fp = fopen(filename.c_str(),"rb");
        while (fp==NULL) {
            std::cerr<<"Tensor:read: fail to open file "<<filename<<". Please provide it first. Will retry after 5 seconds."<<std::endl;
            std::this_thread::sleep_for(std::chrono::seconds(5));
            fp = fopen(filename.c_str(),"rb");
        }
        read(fp,batch_size);
        fclose(fp);
        return CPUmem;
    };

    //write without header
    void writeHeader(FILE* fp, std::vector<int> dim2write){
        uint8_t myTypeid = typeID(typeid(T));
        fwrite((void*)(&myTypeid), sizeof(uint8_t), 1, fp);
        uint32_t typesizeof = uint32_t(sizeof(T));
        fwrite((void*)(&typesizeof), sizeof(uint32_t), 1, fp);
        int lenName = name.size();
        fwrite((void*)(&lenName), sizeof(int), 1, fp);
        if (lenName>0) fwrite((void*)(name.data()), sizeof(char), lenName, fp);
        int nbDims = dim2write.size();
        fwrite((void*)(&nbDims), sizeof(int), 1, fp);
        if (nbDims>0) fwrite((void*)(&dim2write[0]), sizeof(int), nbDims, fp);
        if (ferror (fp)){
            std::cerr << "disk writing failed"<<std::endl;
            FatalError();
        }
    };

    void writeData(FILE* fp, size_t max_size = 0){
        size_t n = numel();
        if (max_size !=0 ) n = min(n,max_size);
        if (n>0){
            fwrite((void*)(CPUmem), sizeof(T), n, fp);
            if (ferror (fp)){
                std::cerr << "disk writing failed" << std::endl;
                FatalError();
            }
        }
    };

    //support continuous write across many NdTensors
    //write with header
    void write(FILE* fp){
        writeHeader(fp,dim);
        writeData(fp);
    };

    void write(std::string filename){
        FILE* fp = fopen(filename.c_str(),"wb");
        while (fp==NULL) {
            std::cerr<<"Tensor::write: fail to open file "<<filename<<". Will retry after 5 seconds."<<std::endl;
            std::this_thread::sleep_for(std::chrono::seconds(5));
            fp = fopen(filename.c_str(),"wb");
        }
        write(fp);
        fclose(fp);
        return;
    };

    Tensor(std::string filename, int batch_size=1): CPUmem(NULL){ read(filename,batch_size); };

    Tensor(FILE* fp): CPUmem(NULL){ read(fp); };

    Tensor(std::vector<int> dim_): dim(dim_){ CPUmem = new T [numel()]; };

    Tensor(std::vector<int> dim_, T initValue): dim(dim_){
        int n = numel();
        CPUmem = new T [n];
        if (initValue == T(0))
            memset(CPUmem, 0, n*sizeof(T));
        else
            for (int i=0;i<n;++i) CPUmem[i] = initValue;

    };

    Tensor(std::string name_, std::vector<int> dim_): name(name_),dim(dim_){
        CPUmem = new T [numel()];
    };

    void permute(std::vector<size_t> v){
        size_t nbItems = numofitems();
        size_t sizeofitem_ = sizeofitem();
        size_t nbBytes = sizeofitem_ * sizeof(T);
        T* CPUmemNew = new T[numel()];
        memcpy(CPUmemNew, CPUmem, nbItems * nbBytes);
        for (size_t i=0;i<nbItems;++i){
            memcpy(CPUmem+i*sizeofitem_, CPUmemNew+v[i]*sizeofitem_, nbBytes);
        }
        delete [] CPUmemNew;
    };


    void printRange(){
        int n = numel();
        if (n==0){
            std::cout<<"Emtpy tensor"<<std::endl;
            return;
        }
        T maxValue = CPUmem[0];
        T minValue = CPUmem[0];

        for (int i=0;i<n;++i){
            if (maxValue<CPUmem[i])     maxValue=CPUmem[i];
            if (CPUmem[i]<minValue)     minValue=CPUmem[i];
        }
        std::cout<< "Value Range ["<<minValue<<", "<<maxValue<<"]"<<std::endl;
    };

    void print(std::vector<int> display_dim){

        std::cout<<"  name:"<<name<<" dim"; veciPrint(dim); std::cout<<std::endl;
        switch (display_dim.size()){
            case 1:
                for (int i=0;i<min((size_t)(display_dim[0]),numel());++i)
                    std::cout<<CPUmem[i]<<" ";
                std::cout<<std::endl;
                break;
            case 2:
                for (int i=0;i<display_dim[0];++i){
                    for (int j=0;j<display_dim[1];++j){
                        std::cout<<(CPUmem[i*dim[display_dim.size()-1]+j])<<" ";
                    }
                    std::cout<<std::endl;
                }
                break;
            case 3:
                for (int i=0;i<display_dim[0];++i){
                    for (int j=0;j<display_dim[1];++j){
                        for (int k=0;k<display_dim[2];++k){
                            std::cout<<CPUmem[i*dim[dim.size()-2]*dim[dim.size()-1]+j*dim[dim.size()-1]+k]<<" ";
                        }
                        std::cout<<std::endl;
                    }
                    std::cout<<std::endl;
                }
                break;
        }

    };
};

//////////////////////////////////////////////////////////////////////////////////////////////////
// CUDA kernels
//////////////////////////////////////////////////////////////////////////////////////////////////


#define CUDA_NUM_THREADS 512

#define MAX_NUM_BLOCKS 2880

inline int CUDA_GET_BLOCKS(const size_t N) {
    return min(MAX_NUM_BLOCKS, int((N + size_t(CUDA_NUM_THREADS) - 1) / CUDA_NUM_THREADS));
}

inline size_t CUDA_GET_LOOPS(const size_t N) {
    size_t total_threads = CUDA_GET_BLOCKS(N)*CUDA_NUM_THREADS;
    return (N + total_threads -1)/ total_threads;
}


__global__ void Kernel_set_value(size_t CUDA_NUM_LOOPS, size_t N, StorageT* GPUdst, StorageT value){
    const size_t idxBase = size_t(CUDA_NUM_LOOPS) * (size_t(CUDA_NUM_THREADS) * size_t(blockIdx.x) + size_t(threadIdx.x));
    if (idxBase >= N) return;
    for (size_t idx = idxBase; idx < min(N,idxBase+CUDA_NUM_LOOPS); ++idx ){
        GPUdst[idx] = value;
    }
}

void GPU_set_value(size_t N, StorageT* GPUdst, StorageT value){
    Kernel_set_value<<<CUDA_GET_BLOCKS(N), CUDA_NUM_THREADS>>>(CUDA_GET_LOOPS(N),N,GPUdst,value);
    checkCUDA(__LINE__,hipGetLastError());
}

void GPU_set_ones(size_t N, StorageT* GPUdst){
    GPU_set_value(N, GPUdst, CPUCompute2StorageT(1));
}

void GPU_set_negones(size_t N, StorageT* GPUdst){
    GPU_set_value(N, GPUdst, CPUCompute2StorageT(-1));
}

void GPU_set_zeros(size_t N, StorageT* GPUdst){
    GPU_set_value(N, GPUdst, CPUCompute2StorageT(0));
}

__global__ void Kernel_integrate(
        unsigned int xSize, unsigned int ySize, unsigned int zSize, 
        ComputeT xMin, ComputeT yMin, ComputeT zMin, 
        ComputeT unit, ComputeT margin, 
        unsigned int width, unsigned int height, 
        const ComputeT* depth, const ComputeT* pose, const ComputeT* intrinsics, StorageT *tsdf, uint8_t *weight) {

    unsigned int x = blockIdx.x;
    unsigned int y = threadIdx.x;

    ComputeT xWorld = xMin + x * unit;
    ComputeT yWorld = yMin + y * unit;
    ComputeT zWorld = zMin;

    ComputeT xCamera = pose[0] * xWorld + pose[1] * yWorld + pose[2] *zWorld + pose[3];
    ComputeT yCamera = pose[4] * xWorld + pose[5] * yWorld + pose[6] *zWorld + pose[7];
    ComputeT zCamera = pose[8] * xWorld + pose[9] * yWorld + pose[10] *zWorld + pose[11];

    ComputeT xDelta = pose[2] * unit;
    ComputeT yDelta = pose[6] * unit;
    ComputeT zDelta = pose[10] * unit;

    unsigned int idx_offset = x * ySize * zSize + y * zSize;

    for (unsigned int z = 0; z < zSize; ++z, xCamera += xDelta, yCamera += yDelta, zCamera += zDelta){

        ComputeT xOzCamera = xCamera / zCamera;
        ComputeT yOzCamera = yCamera / zCamera;

        int px = roundf(intrinsics[0] * xOzCamera + intrinsics[2]);
        int py = roundf(intrinsics[4] * yOzCamera + intrinsics[5]);

        if (px < 0 || px >= width || py < 0 || py >= height) continue;

        ComputeT p_depth = *(depth + py * width + px);

        if (p_depth == 0.0) continue;

        ComputeT diff = (p_depth - zCamera) * sqrtf(1.0 + xOzCamera * xOzCamera + yOzCamera * yOzCamera);

        if(diff > -margin){
            ComputeT v_new = fminf(1.0, diff/margin); //tsdf

            //v_new = 1.0 - fabs(v_new); // 1-tdf // comment this out if you want to use tsdf

            unsigned int idx = idx_offset + z;

            uint8_t w  = weight[idx];
            ComputeT v = GPUStorage2ComputeT(tsdf[idx]);

            tsdf[idx] = GPUCompute2StorageT(fmin(fmax((ComputeT(w)*v + v_new)/(ComputeT(w + 1)), -1.f), 1.f));
            weight[idx] = min(w+1,254);
        }
    }
}

//////////////////////////////////////////////////////////////////////////////////////////////////
// main
//////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv){

    if (argc < 5 || argc >13){
        std::cout<<"Usage:"<<std::endl;
        std::cout<<argv[0]<<" depthMaps.tensor intrinscis.tensor cameraRtW2C.tensor outTSDF.tensor [xMin] [xMax] [yMin] [yMax] [zMin] [zMax] [unit] [margin]"<<std::endl;
        return 0;
    }

    size_t memoryBytes = 0;

    Tensor<ComputeT>* depthMaps_CPU = new Tensor<ComputeT>(argv[1]);
    unsigned int numFrames = depthMaps_CPU->dim[0];
    unsigned int width  = depthMaps_CPU->dim[3];
    unsigned int height = depthMaps_CPU->dim[2];
    std::cout<<"depth maps ["<<numFrames<<", 1, "<<height<<", "<<width<<"]"<<std::endl;

    ComputeT* depthMaps_GPU;
    checkCUDA(__LINE__, hipMalloc(&depthMaps_GPU, depthMaps_CPU->numBytes()) );    memoryBytes+=depthMaps_CPU->numBytes();
    checkCUDA(__LINE__, hipMemcpy(depthMaps_GPU, depthMaps_CPU->CPUmem, depthMaps_CPU->numBytes(), hipMemcpyHostToDevice) );
    delete depthMaps_CPU;


    Tensor<ComputeT>* intrinsics_CPU = new Tensor<ComputeT>(argv[2]);
    ComputeT* intrinsics_GPU;
    checkCUDA(__LINE__, hipMalloc(&intrinsics_GPU, intrinsics_CPU->numBytes()) );    memoryBytes+=intrinsics_CPU->numBytes();
    checkCUDA(__LINE__, hipMemcpy(intrinsics_GPU, intrinsics_CPU->CPUmem, intrinsics_CPU->numBytes(), hipMemcpyHostToDevice) );
    delete intrinsics_CPU;

    Tensor<ComputeT>* cameraRtW2C_CPU = new Tensor<ComputeT>(argv[3]);
    ComputeT* cameraRtW2C_GPU;
    checkCUDA(__LINE__, hipMalloc(&cameraRtW2C_GPU, cameraRtW2C_CPU->numBytes()) );    memoryBytes+=cameraRtW2C_CPU->numBytes();
    checkCUDA(__LINE__, hipMemcpy(cameraRtW2C_GPU, cameraRtW2C_CPU->CPUmem, cameraRtW2C_CPU->numBytes(), hipMemcpyHostToDevice) );
    delete cameraRtW2C_CPU;

    ComputeT xMin;   xMin   = (argc<=5? -0.05 : atof(argv[5] ));     std::cout<<"xMin  ="<<xMin<<std::endl;
    ComputeT xMax;   xMax   = (argc<=6?  0.03 : atof(argv[6] ));     std::cout<<"xMax  ="<<xMax<<std::endl;
    ComputeT yMin;   yMin   = (argc<=7? -0.15 : atof(argv[7] ));     std::cout<<"yMin  ="<<yMin<<std::endl;
    ComputeT yMax;   yMax   = (argc<=8?  0.03 : atof(argv[8] ));     std::cout<<"yMax  ="<<yMax<<std::endl;
    ComputeT zMin;   zMin   = (argc<=9?  0.33 : atof(argv[9] ));     std::cout<<"zMin  ="<<zMin<<std::endl;
    ComputeT zMax;   zMax   = (argc<=10? 0.42 : atof(argv[10]));     std::cout<<"zMax  ="<<zMax<<std::endl;
    ComputeT unit;   unit   = (argc<=11? 0.002 : atof(argv[11]));    std::cout<<"unit  ="<<unit<<std::endl;
    ComputeT margin; margin = (argc<=12? 0.01 : atof(argv[12]));     std::cout<<"margin="<<margin<<std::endl;

    unsigned int xSize = round((xMax-xMin)/unit);
    unsigned int ySize = round((yMax-yMin)/unit);
    unsigned int zSize = round((zMax-zMin)/unit);

    std::cout<<"TSDF resolution: "<<xSize<<"x"<<ySize<<"x"<<zSize<<std::endl;

    StorageT* tsdf_GPU;  checkCUDA(__LINE__, hipMalloc(&tsdf_GPU,  xSize*ySize*zSize*sizeofStorageT) );    memoryBytes+=xSize*ySize*zSize*sizeofStorageT;
    uint8_t* weight_GPU; checkCUDA(__LINE__, hipMalloc(&weight_GPU,xSize*ySize*zSize*sizeof(uint8_t)));    memoryBytes+=xSize*ySize*zSize*sizeof(uint8_t);

    std::cout<<"Total GPU memory: ";    memorySizePrint(memoryBytes); std::cout<<std::endl;

    GPU_set_negones(xSize*ySize*zSize, tsdf_GPU);
    checkCUDA(__LINE__, hipMemset(weight_GPU, 0, sizeof(uint8_t)*xSize*ySize*zSize));

    for (unsigned int f=0;f<numFrames;++f){
        Kernel_integrate<<<xSize,ySize>>>(xSize, ySize, zSize, xMin, yMin, zMin, unit, margin, width, height, 
            depthMaps_GPU+width*height*f, cameraRtW2C_GPU+3*4*f, intrinsics_GPU, tsdf_GPU, weight_GPU);
    }

    std::vector<int> dim;
    dim.push_back(xSize);
    dim.push_back(ySize);
    dim.push_back(zSize);
    Tensor<StorageT>* tsdf_CPU = new Tensor<StorageT>(dim);
    tsdf_CPU->readGPU(tsdf_GPU);
    FILE* fp = fopen(argv[4],"wb");
    tsdf_CPU->write(fp);
    fclose(fp);
    delete tsdf_CPU;  

    if (tsdf_GPU!=NULL) checkCUDA(__LINE__, hipFree(tsdf_GPU));
    if (weight_GPU!=NULL) checkCUDA(__LINE__, hipFree(weight_GPU));
    if (depthMaps_GPU!=NULL) checkCUDA(__LINE__, hipFree(depthMaps_GPU));
    if (cameraRtW2C_GPU!=NULL) checkCUDA(__LINE__, hipFree(cameraRtW2C_GPU));
    if (intrinsics_GPU!=NULL) checkCUDA(__LINE__, hipFree(intrinsics_GPU));

    return 0;
}
